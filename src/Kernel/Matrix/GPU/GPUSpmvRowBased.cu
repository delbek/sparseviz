#include "GPUSpmvRowBased.h"
#include "GPUKernels.cuh"
#include "hip/hip_runtime.h"


bool GPUSpmvRowBased::init(const SparseMatrix &A)
{
    inputVector = new valType[A.getColCount()];
    outputVector = new valType[A.getRowCount()];

    vType inputM = A.getRowCount();

    gpuErrchk( hipMalloc(&deviceM, sizeof(vType)));
    gpuErrchk( hipMemcpy(deviceM, &inputM, sizeof(vType), hipMemcpyHostToDevice));

    gpuErrchk( hipMalloc(&ptrs, (A.getRowCount() + 1) * sizeof(vType)));
    gpuErrchk( hipMemcpy(ptrs, A.getPtr(), (A.getRowCount() + 1) * sizeof(vType), hipMemcpyHostToDevice));

    gpuErrchk( hipMalloc(&inds, A.getNNZCount() * sizeof(vType)));
    gpuErrchk( hipMemcpy(inds, A.getInd(), A.getNNZCount() * sizeof(vType), hipMemcpyHostToDevice));

    gpuErrchk( hipMalloc(&vals, A.getNNZCount() * sizeof(valType)));
    gpuErrchk( hipMemcpy(vals, A.getValues(), A.getNNZCount() * sizeof(valType), hipMemcpyHostToDevice));

    gpuErrchk( hipMalloc(&deviceInput, A.getColCount() * sizeof(valType)));
    gpuErrchk( hipMemcpy(deviceInput, inputVector, A.getColCount() * sizeof(valType), hipMemcpyHostToDevice));
    gpuErrchk( hipMalloc(&deviceOutput, A.getRowCount() * sizeof(valType)));

    return true;
}

GPUSpmvRowBased::~GPUSpmvRowBased()
{
    gpuErrchk( hipFree(deviceM));
    gpuErrchk( hipFree(ptrs));
    gpuErrchk( hipFree(inds));
    gpuErrchk( hipFree(vals));
    gpuErrchk( hipFree(deviceInput));
    gpuErrchk( hipFree(deviceOutput));
    delete[] inputVector;
    delete[] outputVector;
}

void GPUSpmvRowBased::hostFunction(const SparseMatrix &A, int iterNumber, int gridSize, int blockSize)
{
    for(int i = 0; i < 10; i++) {
        spmvRowBased<<<gridSize, blockSize>>>(ptrs, inds, vals, deviceM, deviceInput, deviceOutput);
    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
} 

