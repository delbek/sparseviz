#include "CudaHelloWorld.h"
#include "GPUKernels.cuh"
#include "hip/hip_runtime.h"


bool CudaHelloWorld::init(const SparseMatrix &A)
{
    hostOutput = new char[N];
    return true;
}

CudaHelloWorld::~CudaHelloWorld()
{
    delete[] hostOutput;
}

void CudaHelloWorld::hostFunction(const SparseMatrix &A, int iterNumber, int gridSize, int blockSize)
{
    char* deviceOutput;

    hipMalloc(&deviceOutput, N * sizeof(char));

    cudaHelloWorldKernel<<<gridSize, blockSize>>>(deviceOutput);

    hipMemcpy(hostOutput, deviceOutput, N * sizeof(char), hipMemcpyDeviceToHost);

    std::cout << hostOutput << std::endl;

    hipFree(deviceOutput);
}
